#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;
#include "commonheader.h"
#include "feature.hh"
#include "bkmodel.hh"
#include <stdio.h>
#include "bkmodel_gpu_old.hh"

#define d_pm(i,j) d_pm[NLAT*i+j]
#define d_pu(i,j) d_pu[NLAT*i +j]
#define d_pu1(i,j) d_pu1[NLAT*i+j]
#define d_btm(i,j) d_btm[30*i+j]
#define d_bfm(i,j) d_bfm[8*i+j]
#define PUT_FUNCTION_G \
	for(int i = 0 ; i < NLAT;i++)	\
	{\
		error -= d_pm(im,i) * (d_pu(iu,i) + d_pu1(iu,i)* tt);\
	};\
	error -= MEAN + d_bu[iu] + d_bm[im]  + d_btm(im,it)*d_btu[iu] + d_bt[it] + d_bta[ita] + d_bf[ife]+d_bfm(im,ife);


#define gpuErrchk(ans,i) { gpuAssert((ans), i, __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, int i, const char *file, int line, bool abort=true)
{
	   if (code != hipSuccess) 
		      {
				        fprintf(stderr,"GPUassert: %s %s %d %d\n", hipGetErrorString(code), file, line,i);
						      if (abort) exit(code);
							     }
}

__device__ double atomicAdd(double* address, double val)
{
	    unsigned long long int* address_as_ull = (unsigned long long int*)address;

		    unsigned long long int old = *address_as_ull, assumed;

			    do{ assumed = old;
							old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val +__longlong_as_double(assumed)));
							    } while (assumed != old);

				    return __longlong_as_double(old);
}


__device__ float ug(float a)
{
	if(a>0) return 0.0363636*pow((double)a,0.4);
	else return -0.0363636*pow((double)abs(a),0.4);

}
__global__ void kernel_sgd(
	//model
 	float*d_bu,
 	float*d_bm,
 	float*d_pm,
 	float*d_pu,
 	float*d_pu1,
 	float*d_btu,
 	float*d_bf,
 	float*d_bt,
	float*d_bta,
 	float*d_bfm,
	float*d_btm,
	//data
	int*du,
	int*dm,
	int*dt,
	int*df,
	float*dtt,
	int*rate,
	float _lr,
	int shift,
	int sz
	)
{
	int ith =( shift + threadIdx.x + blockIdx.x * blockDim.x );
	while(ith < sz)
	{
	
	int iu = du[ith];
	int im = dm[ith];
	int it = (int)(dt[ith]/75);
	int ita = dt[ith];
	int ife = df[ith];
//	printf("gpu:%d\t%d\t%d\t%d\t%d\n",iu,im,it,ita,ife);
	float tt =ug(dtt[ith]);
	int ir = rate[ith];
	float tmp;
	float error = (float)ir;
	PUT_FUNCTION_G

	tmp =_lr*(error - 0.008*d_bu[iu]); //no need to regulating thses terms
	atomicAdd(&d_bu[iu],tmp);

	tmp =_lr*(error - 0.008*d_bm[im]);
	atomicAdd(&d_bm[im],tmp);


	for(int i = 0; i < NLAT; i++)
	{
		tmp = _lr*(error * (d_pu(iu,i) + d_pu1(iu,i)*tt) - 0.015 * d_pm(im,i)) ;
		atomicAdd(&(d_pm(im,i)),tmp);

		tmp = _lr*(error * d_pm(im,i) -0.015 * d_pu(iu,i));
		atomicAdd(&(d_pu(iu,i)),tmp);
	//	tmp = 0.5*_lr*(error*tt*d_pm(im,i)-0.015*d_pu1(iu,i));//		}
// step3 	
		tmp =1.0*_lr*(error*tt*d_pm(im,i) - 0.015*d_pu1(iu,i));//		}
		atomicAdd(&(d_pu1(iu,i)),tmp);

	}
	tmp = _lr*(error*d_btu[iu] - 0.015 * d_btm(im,it));
	atomicAdd(&(d_btm(im,it)), tmp);
	tmp = _lr*(error*d_btm(im,it) - 0.015* d_btu[iu]);
	atomicAdd(&(d_btu[iu]), tmp);
	tmp = 0.1*_lr*( error - 0.008 * d_bt[it]);
	atomicAdd(&(d_bt[it]), tmp);
	tmp = 0.5*_lr*(error - 0.008 * d_bf[ife]);
	atomicAdd(&(d_bf[ife]), tmp);

	tmp = _lr*(error - 0.008 * d_bfm(im,ife));
	atomicAdd(&(d_bfm(im,ife)),tmp);
	tmp = _lr*(error - 0.008 * d_bta[ita]);
	atomicAdd(&(d_bta[ita]),tmp);
	ith += blockDim.x * gridDim.x;
	__syncthreads();
}
	
//	atomicAdd(d_mean, 0.002*error);
//

	return;
}

__global__ void kernel_rmse(
	//model
 	float*d_bu,
 	float*d_bm,
 	float*d_pm,
 	float*d_pu,
 	float*d_pu1,
 	float*d_btu,
 	float*d_bf,
 	float*d_bt,
	float*d_bta,
 	float*d_bfm,
	float*d_btm,
	//data
	int*du,
	int*dm,
	int*dt,
	int*df,
	float*dtt,
	int*rate,
	int sz,
	double *sum
	)
{
	int ith =( blockIdx.x * blockDim.x + threadIdx.x );
	
	while(ith < sz)
	{

		int iu = du[ith];
		int im = dm[ith];
		int it = (int)(dt[ith]/75);
		int ita = dt[ith];
		int ife = df[ith];
//	printf("%d\n",ith);
		float tt =ug(dtt[ith]);
		int ir = rate[ith];
		double error = (double)ir;

		PUT_FUNCTION_G
//		printf("gpu:%d\t %f\n",iu, error);

		atomicAdd(sum, (1.0*error*error));
		ith += gridDim.x* blockDim.x;
//
	}

	return;
}



template<typename T>
void copying_to_gpu1d(T** a, vector<T>&b,int err = 0)
{
	gpuErrchk(hipMalloc((void**)a, sizeof(T)*b.size()),err);
	gpuErrchk(hipMemcpy(*a,&(b[0]), sizeof(T)*b.size(),hipMemcpyHostToDevice),err);
}

template<typename T>
void copying_to_gpu2d(T** a, vector<vector<T>> &b, int err=0)
{
	gpuErrchk(hipMalloc((void**)a, sizeof(T)*b[0].size()*b.size()), err);
	auto it = b.begin();
	T* p = *a;
	while(it != b.end())
	{
		T* src = &((*it)[0]);
		size_t sz = it->size();
		gpuErrchk(hipMemcpy(p, src, sizeof(T)*sz, hipMemcpyHostToDevice), err);
		p+= sz;
		it++;
	}
}

template<typename T>
void copying_to_cpu1d(T** a, vector<T>&b, int err=0)
{
	cout << b.size() << endl;
	gpuErrchk(hipMemcpy(&(b[0]),*a, sizeof(T)*b.size(),hipMemcpyDeviceToHost), err);
}

template<typename T>
void copying_to_cpu2d(T** a, vector<vector<T>>& b, int err=0)
{
	auto it = b.begin();
	T* p = *a;
	while(it != b.end())
	{
		T* src = &((*it)[0]);
		size_t sz = it->size();
		gpuErrchk(hipMemcpy(src, p, sizeof(T)*sz, hipMemcpyDeviceToHost),err);
		p+= sz;
		it++;
	}
}
bkmodel_gpu::bkmodel_gpu():bkmodel()
{
	copying_to_gpu1d(&d_bu,bu);
	copying_to_gpu1d(&d_btu,btu);
	copying_to_gpu1d(&d_bt,bt);
	copying_to_gpu1d(&d_bf,bf);
	copying_to_gpu1d(&d_bm,bm);
	copying_to_gpu1d(&d_bta,bta);
	copying_to_gpu2d(&d_pm,pm);
	copying_to_gpu2d(&d_pu,pu);
	copying_to_gpu2d(&d_pu1,pu1);
	copying_to_gpu2d(&d_bfm, bfm);
	copying_to_gpu2d(&d_btm, btm);

}
void bkmodel_gpu::retrieve_gpu()
{
	copying_to_cpu1d(&d_bu,bu,1001);
	copying_to_cpu1d(&d_btu,btu,1002);
	copying_to_cpu1d(&d_bt,bt,1003);
	copying_to_cpu1d(&d_bf,bf,1004);
	copying_to_cpu1d(&d_bm,bm,1005);
	copying_to_cpu1d(&d_bta,bta,1011);
	copying_to_cpu2d(&d_pm,pm,1006);
	copying_to_cpu2d(&d_pu,pu,1007);
	copying_to_cpu2d(&d_pu1,pu1,1008);
	copying_to_cpu2d(&d_bfm, bfm,1009);
	copying_to_cpu2d(&d_btm, btm,1010);
}

void bkmodel_gpu::loaddata(feature &a0, feature &a1, feature &a2)
{
	sz0 = a0.viu.size();
	copying_to_gpu1d(&d_iu,a0.viu);
	copying_to_gpu1d(&d_im,a0.vim);
	copying_to_gpu1d(&d_it,a0.vita); //ita is from 1-2243
	copying_to_gpu1d(&d_if,a0.vif);
	copying_to_gpu1d(&d_tb,a0.vtb);
	copying_to_gpu1d(&d_rate,a0.vrate);

	sz1 = a1.viu.size();
	copying_to_gpu1d(&d_iu1,a1.viu);
	copying_to_gpu1d(&d_im1,a1.vim);
	copying_to_gpu1d(&d_it1,a1.vita);
	copying_to_gpu1d(&d_if1,a1.vif);
	copying_to_gpu1d(&d_tb1,a1.vtb);
	copying_to_gpu1d(&d_rate1,a1.vrate);

	sz2 = a2.viu.size();
	copying_to_gpu1d(&d_iu2,a2.viu);
	copying_to_gpu1d(&d_im2,a2.vim);
	copying_to_gpu1d(&d_it2,a2.vita);
	copying_to_gpu1d(&d_if2,a2.vif);
	copying_to_gpu1d(&d_tb2,a2.vtb);
	copying_to_gpu1d(&d_rate2,a2.vrate);

}
double bkmodel_gpu::compute_error()
{
	double sum1 = 0, *dev_sum1;
	hipMalloc((void**) &dev_sum1,sizeof(double));
	hipMemcpy(dev_sum1, &sum1, sizeof(double), hipMemcpyHostToDevice);

	double sum2 = 0, *dev_sum2;
	hipMalloc((void**) &dev_sum2,sizeof(double));
	hipMemcpy(dev_sum2, &sum2, sizeof(double), hipMemcpyHostToDevice);

	kernel_rmse<<<32,32>>>(
	//model
	d_bu,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
 	d_btu,
 	d_bf,
 	d_bt,
	d_bta,
 	d_bfm,
 	d_btm,
	//data
	d_iu1,
	d_im1,
	d_it1,
	d_if1,
	d_tb1,
	d_rate1,
	sz1,
	dev_sum1);
	gpuErrchk(hipPeekAtLastError(),400);

	kernel_rmse<<<32,32>>>(
	//model
	d_bu,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
 	d_btu,
 	d_bf,
 	d_bt,
	d_bta,
 	d_bfm,
 	d_btm,
	//data
	d_iu2,
	d_im2,
	d_it2,
	d_if2,
	d_tb2,
	d_rate2,
	sz2,
	dev_sum2);
	gpuErrchk(hipPeekAtLastError(),400)
	hipMemcpy( &sum1, dev_sum1,sizeof(double),  hipMemcpyDeviceToHost);
	hipMemcpy( &sum2, dev_sum2,sizeof(double),  hipMemcpyDeviceToHost);
	cout << "in sample error: " << sqrt(sum1/sz1) << endl;
	cout << "out sample error: " <<sqrt(sum2/sz2) << endl;
	double rmse = sqrt(sum2/sz2);

	return rmse;


}

void bkmodel_gpu::test(float lr)
{
	kernel_sgd<<<32,32>>>(
	//model
	d_bu,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
 	d_btu,
 	d_bf,
 	d_bt,
	d_bta,
 	d_bfm,
 	d_btm,
	//data
	d_iu,
	d_im,
	d_it,
	d_if,
	d_tb,
	d_rate,
	lr,
	0,
	sz0);
	gpuErrchk(hipPeekAtLastError(),300);



/*	testgpu<<<16,16>>>(d_bm,d_bt);
:	hipMemcpy(a,d_bm, 16*sizeof(float),hipMemcpyDeviceToHost);
	cout << a[10] <<"\t"<< a[11] <<"\t"<< a[12] <<endl;
	cout << bm[1][0] << "\t"<< bm[1][1] << "\t" << bm[1][2] << endl;
	gpuErrchk(hipPeekAtLastError());
*/}
