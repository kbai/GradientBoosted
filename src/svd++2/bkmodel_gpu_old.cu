#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;
#include "commonheader.h"
#include "feature.hh"
#include "bkmodel.hh"
#include <stdio.h>
#include "bkmodel_gpu_old.hh"

#define d_pm(i,j) d_pm[NLAT*i+j]
#define d_pu(i,j) d_pu[NLAT*i +j]
#define d_pu1(i,j) d_pu1[NLAT*i+j]
#define d_ptu(i,j) d_ptu[NLARGE*(int)(j)+i]
#define d_btm(i,j) d_btm[30*i+j]
#define d_bfm(i,j) d_bfm[8*i+j]
#define ETA_BU 0.003
#define LAMBDA_BU 0.03
#define ETA_BUT 0.0025
#define LAMBDA_BUT 0.005
#define ETA_AU 0.00001
#define LAMBDA_AU 50
#define ETA_BM 0.002
#define LAMBDA_BM 0.03
#define ETA_BMT 0.00005
#define LAMBDA_BMT 0.1
#define ETA_CU 0.008
#define LAMBDA_CU 0.01
#define ETA_CUT 0.002
#define LAMBDA_CUT 0.005
#define ETA_PU 0.008
#define LAMBDA_PU 0.015
#define ETA_PM 0.008
#define LAMBDA_PM 0.015
#define ETA_AL 1E-5
#define LAMBDA_AL 50

#define PUT_FUNCTION_G \
	for(int i = 0 ; i < NLAT;i++)	\
	{\
		error -= d_pm(im,i) * (d_pu(iu,i) + d_pu1(iu,i) * (tt+d_htu[itu])+ d_ptu(itu,i/10));\
	};\
	error -= MEAN + d_au[iu]*tt + d_bu[iu] + d_but[itu] + (d_bm[im] + d_btm(im,it))*(d_btu[iu] + d_cut[itu])\
		+ d_bt[it] + d_bta[ita] + d_bf[ife]+d_bfm(im,ife);


#define gpuErrchk(ans,i) { gpuAssert((ans), i, __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, int i, const char *file, int line, bool abort=true)
{
	   if (code != hipSuccess) 
		      {
				        fprintf(stderr,"GPUassert: %s %s %d %d\n", hipGetErrorString(code), file, line,i);
						      if (abort) exit(code);
							     }
}

__device__ double atomicAdd(double* address, double val)
{
	    unsigned long long int* address_as_ull = (unsigned long long int*)address;

		    unsigned long long int old = *address_as_ull, assumed;

			    do{ assumed = old;
							old = atomicCAS(address_as_ull, assumed,__double_as_longlong(val +__longlong_as_double(assumed)));
							    } while (assumed != old);

				    return __longlong_as_double(old);
}


__device__ float ug(float a)
{
	if(a>0) return pow((double)a,0.4);
	else return -pow((double)abs(a),0.4);

}
__global__ void kernel_sgd(
	//model
 	float*d_bu,
	float*d_au,
 	float*d_bm,
 	float*d_pm,
 	float*d_pu,
 	float*d_pu1,
	float*d_ptu,
 	float*d_btu,
 	float*d_bf,
 	float*d_bt,
	float*d_bta,
 	float*d_bfm,
	float*d_btm,
	float*d_htu,
	float*d_cut,
	float*d_but,
	//data
	int*du,
	int*dm,
	int*dt,
	int*df,
	float*dtt,
	int*dtu,
	int*rate,
	float _lr,
	int shift,
	int sz
	)
{
	int ith =( shift + threadIdx.x + blockIdx.x * blockDim.x );
	while(ith < sz)
	{
	
	int iu = du[ith];
	int im = dm[ith];
	int it = (int)(dt[ith]/75);
	int ita = dt[ith];
	int ife = df[ith];
	int itu = dtu[ith];
//	printf("gpu:%d\t%d\t%d\t%d\t%d\n",iu,im,it,ita,ife);
	float tt =ug(dtt[ith]);
	int ir = rate[ith];
	float tmp;
	float error = (float)ir;
	PUT_FUNCTION_G

	tmp = _lr * ETA_BU * (error - LAMBDA_BU * d_bu[iu]); //no need to regulating thses terms
	atomicAdd(&d_bu[iu],tmp);

	tmp = _lr * ETA_AU * (error*tt - LAMBDA_AU * d_au[iu]);
	atomicAdd(&d_au[iu],tmp);

	tmp = _lr * ETA_BUT * (error - LAMBDA_BUT * d_but[itu]);
	atomicAdd(&d_but[itu], tmp);

	tmp = _lr * ETA_BM * (error * (d_btu[iu] + d_cut[itu]) - LAMBDA_BM * d_bm[im]);
	atomicAdd(&d_bm[im],tmp);

	tmp = _lr * ETA_BMT *(error * (d_btu[iu] + d_cut[iu]) - LAMBDA_BMT * d_btm(im,it));
	atomicAdd(&(d_btm(im,it)), tmp);

	tmp = _lr * ETA_CU *(error * (d_btm(im,it) + d_bm[im]) - LAMBDA_CU * (d_btu[iu] - 1));
	atomicAdd(&(d_btu[iu]), tmp);

	tmp = _lr * ETA_CUT *(error * (d_btm(im,it) + d_bm[im]) - LAMBDA_CUT * (d_cut[itu]));
	atomicAdd(&(d_cut[itu]), tmp);

	for(int i = 0; i < NLAT; i++)
	{
		tmp = _lr * ETA_PM * (error * (d_pu(iu,i) + d_pu1(iu,i)*( tt + d_htu[itu] + d_ptu(itu,i/10) ) ) - LAMBDA_PM * d_pm(im,i)) ;
		atomicAdd(&(d_pm(im,i)),tmp);

		tmp = _lr * ETA_PU * (error * d_pm(im,i) -LAMBDA_PU * d_pu(iu,i));
		atomicAdd(&(d_pu(iu,i)),tmp);

//		tmp = _lr * 0.1* ETA_PU * (error * d_pm(im,i) - LAMBDA_PU * d_ptu(itu,i/10));
//		atomicAdd(&(d_ptu(itu,i/10)),tmp);


//		tmp = _lr * ETA_PU * (error*d_pm(im,i)*d_pu1(iu,i) - LAMBDA_PU * d_htu[itu])/NLAT;
//		atomicAdd(&(d_htu[itu]),tmp);


		tmp = _lr * ETA_AL * (error*(tt + d_htu[itu])*d_pm(im,i) - LAMBDA_AL * d_pu1(iu,i));//		}
		atomicAdd(&(d_pu1(iu,i)),tmp);

	}

/*
	tmp = 0.1*_lr*( error - 0.008 * d_bt[it]);
	atomicAdd(&(d_bt[it]), tmp);
	tmp = 0.01*_lr*(error - 0.008 * d_bf[ife]);
	atomicAdd(&(d_bf[ife]), tmp);

	tmp = _lr*(error - 0.008 * d_bfm(im,ife));
	atomicAdd(&(d_bfm(im,ife)),tmp);
	tmp = _lr*(error - 0.008 * d_bta[ita]);
	atomicAdd(&(d_bta[ita]),tmp);
*/
	ith += blockDim.x * gridDim.x;
	__syncthreads();
}
	
//	atomicAdd(d_mean, 0.002*error);
//

	return;
}

__global__ void kernel_rmse(
	//model
 	float*d_bu,
	float*d_au,
 	float*d_bm,
 	float*d_pm,
 	float*d_pu,
 	float*d_pu1,
	float*d_ptu,
 	float*d_btu,
 	float*d_bf,
 	float*d_bt,
	float*d_bta,
 	float*d_bfm,
	float*d_btm,
	float*d_htu,
	float*d_cut,
	float*d_but,
	//data
	int*du,
	int*dm,
	int*dt,
	int*df,
	float*dtt,
	int*dtu,
	int*rate,
	int sz,
	double *sum
	)
{
	int ith =( blockIdx.x * blockDim.x + threadIdx.x );
	
	while(ith < sz)
	{

		int iu = du[ith];
		int im = dm[ith];
		int it = (int)(dt[ith]/75);
		int ita = dt[ith];
		int ife = df[ith];
		int itu = dtu[ith];
//	printf("%d\n",ith);
		float tt =ug(dtt[ith]);
		int ir = rate[ith];
		double error = (double)ir;

		PUT_FUNCTION_G
//		printf("gpu:%d\t %f\n",iu, error);

		atomicAdd(sum, (1.0*error*error));
		ith += gridDim.x* blockDim.x;
//
	}

	return;
}



template<typename T>
void copying_to_gpu1d(T** a, vector<T>&b,int err = 0)
{
	gpuErrchk(hipMalloc((void**)a, sizeof(T)*b.size()),err);
	gpuErrchk(hipMemcpy(*a,&(b[0]), sizeof(T)*b.size(),hipMemcpyHostToDevice),err);
}

template<typename T>
void copying_to_gpu2d(T** a, vector<vector<T>> &b, int err=0)
{
	gpuErrchk(hipMalloc((void**)a, sizeof(T)*b[0].size()*b.size()), err);
	auto it = b.begin();
	T* p = *a;
	while(it != b.end())
	{
		T* src = &((*it)[0]);
		size_t sz = it->size();
		gpuErrchk(hipMemcpy(p, src, sizeof(T)*sz, hipMemcpyHostToDevice), err);
		p+= sz;
		it++;
	}
}

template<typename T>
void copying_to_cpu1d(T** a, vector<T>&b, int err=0)
{
	cout << b.size() << endl;
	gpuErrchk(hipMemcpy(&(b[0]),*a, sizeof(T)*b.size(),hipMemcpyDeviceToHost), err);
}

template<typename T>
void copying_to_cpu2d(T** a, vector<vector<T>>& b, int err=0)
{
	auto it = b.begin();
	T* p = *a;
	while(it != b.end())
	{
		T* src = &((*it)[0]);
		size_t sz = it->size();
		gpuErrchk(hipMemcpy(src, p, sizeof(T)*sz, hipMemcpyDeviceToHost),err);
		p+= sz;
		it++;
	}
}
bkmodel_gpu::bkmodel_gpu():bkmodel()
{
	copying_to_gpu1d(&d_bu,bu);
	copying_to_gpu1d(&d_btu,btu);
	copying_to_gpu1d(&d_bt,bt);
	copying_to_gpu1d(&d_bf,bf);
	copying_to_gpu1d(&d_bm,bm);
	copying_to_gpu1d(&d_bta,bta);
	copying_to_gpu2d(&d_pm,pm);
	copying_to_gpu2d(&d_pu,pu);
	copying_to_gpu2d(&d_pu1,pu1);
	copying_to_gpu2d(&d_bfm, bfm);
	copying_to_gpu2d(&d_btm, btm);
	copying_to_gpu1d(&d_htu, htu);
	copying_to_gpu2d(&d_ptu, ptu);
	copying_to_gpu1d(&d_au, au);
	copying_to_gpu1d(&d_but, but);
	copying_to_gpu1d(&d_cut, cut);


}
void bkmodel_gpu::retrieve_gpu()
{
	copying_to_cpu1d(&d_bu,bu,1001);
	copying_to_cpu1d(&d_btu,btu,1002);
	copying_to_cpu1d(&d_bt,bt,1003);
	copying_to_cpu1d(&d_bf,bf,1004);
	copying_to_cpu1d(&d_bm,bm,1005);
	copying_to_cpu1d(&d_bta,bta,1011);
	copying_to_cpu2d(&d_pm,pm,1006);
	copying_to_cpu2d(&d_pu,pu,1007);
	copying_to_cpu2d(&d_pu1,pu1,1008);
	copying_to_cpu2d(&d_bfm, bfm,1009);
	copying_to_cpu2d(&d_btm, btm,1010);
	copying_to_cpu2d(&d_ptu, ptu,1013);
	copying_to_cpu1d(&d_htu, htu,1011);
	copying_to_cpu1d(&d_au, au, 1012);
	copying_to_cpu1d(&d_but, but, 1014);
	copying_to_cpu1d(&d_cut, cut, 1015);
}

void bkmodel_gpu::loaddata(feature &a0, feature &a1, feature &a2)
{
	sz0 = a0.viu.size();
	copying_to_gpu1d(&d_iu,a0.viu);
	copying_to_gpu1d(&d_im,a0.vim);
	copying_to_gpu1d(&d_it,a0.vita); //ita is from 1-2243
	copying_to_gpu1d(&d_if,a0.vif);
	copying_to_gpu1d(&d_tb,a0.vtb);
	copying_to_gpu1d(&d_rate,a0.vrate);
	copying_to_gpu1d(&d_tu, a0.vtut);

	sz1 = a1.viu.size();
	copying_to_gpu1d(&d_iu1,a1.viu);
	copying_to_gpu1d(&d_im1,a1.vim);
	copying_to_gpu1d(&d_it1,a1.vita);
	copying_to_gpu1d(&d_if1,a1.vif);
	copying_to_gpu1d(&d_tb1,a1.vtb);
	copying_to_gpu1d(&d_rate1,a1.vrate);
	copying_to_gpu1d(&d_tu1, a1.vtut);


	sz2 = a2.viu.size();
	copying_to_gpu1d(&d_iu2,a2.viu);
	copying_to_gpu1d(&d_im2,a2.vim);
	copying_to_gpu1d(&d_it2,a2.vita);
	copying_to_gpu1d(&d_if2,a2.vif);
	copying_to_gpu1d(&d_tb2,a2.vtb);
	copying_to_gpu1d(&d_rate2,a2.vrate);
	copying_to_gpu1d(&d_tu2, a2.vtut);


}
double bkmodel_gpu::compute_error()
{
	double sum1 = 0, *dev_sum1;
	hipMalloc((void**) &dev_sum1,sizeof(double));
	hipMemcpy(dev_sum1, &sum1, sizeof(double), hipMemcpyHostToDevice);

	double sum2 = 0, *dev_sum2;
	hipMalloc((void**) &dev_sum2,sizeof(double));
	hipMemcpy(dev_sum2, &sum2, sizeof(double), hipMemcpyHostToDevice);

	kernel_rmse<<<32,32>>>(
	//model
	d_bu,
	d_au,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
	d_ptu,
 	d_btu,
 	d_bf,
 	d_bt,
	d_bta,
 	d_bfm,
 	d_btm,
	d_htu,
	d_cut,
	d_but,
	//data
	d_iu1,
	d_im1,
	d_it1,
	d_if1,
	d_tb1,
	d_tu1,
	d_rate1,
	sz1,
	dev_sum1);
	gpuErrchk(hipPeekAtLastError(),400);

	kernel_rmse<<<32,32>>>(
	//model
	d_bu,
	d_au,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
	d_ptu,
 	d_btu,
 	d_bf,
 	d_bt,
	d_bta,
 	d_bfm,
 	d_btm,
	d_htu,
	d_cut,
	d_but,
	//data
	d_iu2,
	d_im2,
	d_it2,
	d_if2,
	d_tb2,
	d_tu2,
	d_rate2,
	sz2,
	dev_sum2);
	gpuErrchk(hipPeekAtLastError(),400)
	hipMemcpy( &sum1, dev_sum1,sizeof(double),  hipMemcpyDeviceToHost);
	hipMemcpy( &sum2, dev_sum2,sizeof(double),  hipMemcpyDeviceToHost);
	cout << "in sample error: " << sqrt(sum1/sz1) << endl;
	cout << "out sample error: " <<sqrt(sum2/sz2) << endl;
	double rmse = sqrt(sum2/sz2);

	return rmse;


}

void bkmodel_gpu::test(float lr)
{
	kernel_sgd<<<32,32>>>(
	//model
	d_bu,
	d_au,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
	d_ptu,
 	d_btu,
 	d_bf,
 	d_bt,
	d_bta,
 	d_bfm,
 	d_btm,
	d_htu,
	d_cut,
	d_but,
	//data
	d_iu,
	d_im,
	d_it,
	d_if,
	d_tb,
	d_tu,
	d_rate,
	lr,
	0,
	sz0);
	gpuErrchk(hipPeekAtLastError(),300);



/*	testgpu<<<16,16>>>(d_bm,d_bt);
:	hipMemcpy(a,d_bm, 16*sizeof(float),hipMemcpyDeviceToHost);
	cout << a[10] <<"\t"<< a[11] <<"\t"<< a[12] <<endl;
	cout << bm[1][0] << "\t"<< bm[1][1] << "\t" << bm[1][2] << endl;
	gpuErrchk(hipPeekAtLastError());
*/}
