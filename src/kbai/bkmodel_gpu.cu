#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;
#include "commonheader.h"
#include "feature.hh"
#include "bkmodel.hh"
#include "bkmodel_gpu.hh"
#define NLAT 10

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	   if (code != hipSuccess) 
		      {
				        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
						      if (abort) exit(code);
							     }
}




__global__ void testgpu(float * a, float* b)
{
	int it = threadIdx.x + 16 * threadIdx.y;
}




template<typename T>
void copying_to_gpu1d(T** a, vector<T>b)
{
	gpuErrchk(hipMalloc((void**)a, sizeof(T)*b.size()));
	gpuErrchk(hipMemcpy(*a,&(b[0]), sizeof(T)*b.size(),hipMemcpyHostToDevice));
}

template<typename T>
void copying_to_gpu2d(T** a, vector<vector<T>> b)
{
	gpuErrchk(hipMalloc((void**)a, sizeof(T)*b[0].size()*b.size()));
	auto it = b.begin();
	T* p = *a;
	while(it != b.end())
	{
		T* src = &((*it)[0]);
		size_t sz = it->size();
		gpuErrchk(hipMemcpy(p, src, sizeof(T)*sz, hipMemcpyHostToDevice));
		p+= sz;
		it++;
	}
}

bkmodel_gpu::bkmodel_gpu():bkmodel()
{
	copying_to_gpu1d(&d_bu,bu);
	copying_to_gpu1d(&d_bt,bt);
	copying_to_gpu1d(&d_bm,bm);
	copying_to_gpu2d(&d_btm,btm);
	copying_to_gpu1d(&d_btu,btu);
		
}

void bkmodel_gpu::test()
{
	float a[16];
/*	testgpu<<<16,16>>>(d_bm,d_bt);
	hipMemcpy(a,d_bm, 16*sizeof(float),hipMemcpyDeviceToHost);
	cout << a[10] <<"\t"<< a[11] <<"\t"<< a[12] <<endl;
	cout << bm[1][0] << "\t"<< bm[1][1] << "\t" << bm[1][2] << endl;
	gpuErrchk(hipPeekAtLastError());
*/}
