#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
using namespace std;
#include "commonheader.h"
#include "feature.hh"
#include "bkmodel.hh"
#include <stdio.h>
#include "bkmodel_gpu.hh"

#define d_pm(i,j) d_pm[NLAT*i+j]
#define d_pu(i,j) d_pu[NLAT*i +j]
#define d_pu1(i,j) d_pu1[NLAT*i+j]
#define d_btm(i,j) d_btm[30*i+j]
#define d_bfm(i,j) d_bfm[8*i+j]

#define gpuErrchk(ans,i) { gpuAssert((ans), i, __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, int i, const char *file, int line, bool abort=true)
{
	   if (code != hipSuccess) 
		      {
				        fprintf(stderr,"GPUassert: %s %s %d %d\n", hipGetErrorString(code), file, line,i);
						      if (abort) exit(code);
							     }
}


__device__ float ug(float a)
{
	if(a>0) return pow((double)a,0.4);
	else return -pow((double)abs(a),0.4);

}
__global__ void kernel_sgd(
	//model
 	float*d_bu,
 	float*d_bm,
 	float*d_pm,
 	float*d_pu,
 	float*d_pu1,
 	float*d_btu,
 	float*d_bf,
 	float*d_bt,
 	float*d_bfm,
	float*d_btm,
	//data
	int*du,
	int*dm,
	int*dt,
	int*df,
	float*dtt,
	int*rate,
	float _lr,
	int shift,
	int sz
	)
{
	int ith =( shift + threadIdx.x + blockIdx.x * blockDim.x ) % sz;
	
	int iu = du[ith];
	int im = dm[ith];
	int it = dt[ith];
	int ife = df[ith];
//	printf("%d\n",ith);
	float tt =ug(dtt[ith]);
	int ir = rate[ith];
	float tmp;
	float error = (float)ir;
	for(int i = 0 ; i < NLAT;i++)	
	{
		error -= d_pm(im,i) * (d_pu(iu,i) + d_pu1(iu,i)* tt);
	};
//	printf("%d,%d,%d,%d\n",iu,im,it,ife);

	error -= 3.6095162 + d_bu[iu] + d_bm[im]  + d_btm(im,it)*d_btu[iu] + d_bt[it] + d_bf[ife]+d_bfm(im,ife);
//	printf("error: %f\n",error);

	tmp =_lr*(error - 0.008*d_bu[iu]); //no need to regulating thses terms
	atomicAdd(&d_bu[iu],tmp);

	tmp =_lr*(error - 0.008*d_bm[im]);
	atomicAdd(&d_bm[im],tmp);


	for(int i = 0; i < NLAT; i++)
	{
		tmp = _lr*(error * (d_pu(iu,i) + d_pu1(iu,i)*tt) - 0.015 * d_pm(im,i)) ;
		atomicAdd(&(d_pm(im,i)),tmp);

		tmp = _lr*(error * d_pm(im,i) -0.015 * d_pu(iu,i));
		atomicAdd(&(d_pu(iu,i)),tmp);

		tmp = 0.001*_lr*(error*tt*d_pm(im,i) - 0.015*d_pu1(iu,i));//		}
		atomicAdd(&(d_pu1(iu,i)),tmp);

	}
	tmp = _lr*(error*d_btu[iu] - 0.015 * d_btm(im,it));
	atomicAdd(&(d_btm(im,it)), tmp);
	tmp = _lr*(error*d_btm(im,it) - 0.015* d_btu[iu]);
	atomicAdd(&(d_btu[iu]), tmp);
	tmp = _lr*( error - 0.008 * d_bt[it]);
	atomicAdd(&(d_bt[it]), tmp);
	tmp = 0.01*_lr*(error - 0.008 * d_bf[ife]);
	atomicAdd(&(d_bf[ife]), tmp);
/*
	tmp = _lr*(error - 0.008 * d_bfm(im,ife));
	atomicAdd(&(d_bfm(im,ife)),tmp);
	*/
//	atomicAdd(d_mean, 0.002*error);
//

	return;
}

__global__ void kernel_rmse(
	//model
 	float*d_bu,
 	float*d_bm,
 	float*d_pm,
 	float*d_pu,
 	float*d_pu1,
 	float*d_btu,
 	float*d_bf,
 	float*d_bt,
 	float*d_bfm,
	float*d_btm,
	//data
	int*du,
	int*dm,
	int*dt,
	int*df,
	float*dtt,
	int*rate,
	int sz,
	float *sum
	)
{


	int ith =( blockIdx.x * blockDim.x + threadIdx.x );
	
	while(ith < sz)
	{

		int iu = du[ith];
		int im = dm[ith];
		int it = dt[ith];
		int ife = df[ith];
//	printf("%d\n",ith);
		float tt =ug(dtt[ith]);
		int ir = rate[ith];
		float error = (float)ir;
		for(int i = 0 ; i < NLAT;i++)	
		{
			error -= d_pm(im,i) * (d_pu(iu,i) + d_pu1(iu,i)* tt);
		};
//	printf("%d,%d,%d,%d\n",iu,im,it,ife);


		error -= 3.6095162 + d_bu[iu] + d_bm[im]  + d_btm(im,it)*d_btu[iu] + d_bt[it] + d_bf[ife]+d_bfm(im,ife);
//	printf("error: %f\n",error);

		atomicAdd(sum, 1.0*error*error);
		ith += gridDim.x* blockDim.x;
//
	}

	return;
}



template<typename T>
void copying_to_gpu1d(T** a, vector<T>&b,int err = 0)
{
	gpuErrchk(hipMalloc((void**)a, sizeof(T)*b.size()),err);
	gpuErrchk(hipMemcpy(*a,&(b[0]), sizeof(T)*b.size(),hipMemcpyHostToDevice),err);
}

template<typename T>
void copying_to_gpu2d(T** a, vector<vector<T>> &b, int err=0)
{
	gpuErrchk(hipMalloc((void**)a, sizeof(T)*b[0].size()*b.size()), err);
	auto it = b.begin();
	T* p = *a;
	while(it != b.end())
	{
		T* src = &((*it)[0]);
		size_t sz = it->size();
		gpuErrchk(hipMemcpy(p, src, sizeof(T)*sz, hipMemcpyHostToDevice), err);
		p+= sz;
		it++;
	}
}

template<typename T>
void copying_to_cpu1d(T** a, vector<T>&b, int err=0)
{
	cout << b.size() << endl;
	gpuErrchk(hipMemcpy(&(b[0]),*a, sizeof(T)*b.size(),hipMemcpyDeviceToHost), err);
}

template<typename T>
void copying_to_cpu2d(T** a, vector<vector<T>>& b, int err=0)
{
	auto it = b.begin();
	T* p = *a;
	while(it != b.end())
	{
		T* src = &((*it)[0]);
		size_t sz = it->size();
		gpuErrchk(hipMemcpy(src, p, sizeof(T)*sz, hipMemcpyDeviceToHost),err);
		p+= sz;
		it++;
	}
}
bkmodel_gpu::bkmodel_gpu():bkmodel()
{
	copying_to_gpu1d(&d_bu,bu);
	copying_to_gpu1d(&d_btu,btu);
	copying_to_gpu1d(&d_bt,bt);
	copying_to_gpu1d(&d_bf,bf);
	copying_to_gpu1d(&d_bm,bm);
	copying_to_gpu2d(&d_pm,pm);
	copying_to_gpu2d(&d_pu,pu);
	copying_to_gpu2d(&d_pu1,pu1);
	copying_to_gpu2d(&d_bfm, bfm);
	copying_to_gpu2d(&d_btm, btm);

}
void bkmodel_gpu::retrieve_gpu()
{
	copying_to_cpu1d(&d_bu,bu,1001);
	copying_to_cpu1d(&d_btu,btu,1002);
	copying_to_cpu1d(&d_bt,bt,1003);
	copying_to_cpu1d(&d_bf,bf,1004);
	copying_to_cpu1d(&d_bm,bm,1005);
	copying_to_cpu2d(&d_pm,pm,1006);
	copying_to_cpu2d(&d_pu,pu,1007);
	copying_to_cpu2d(&d_pu1,pu1,1008);
	copying_to_cpu2d(&d_bfm, bfm,1009);
	copying_to_cpu2d(&d_btm, btm,1010);
}

void bkmodel_gpu::loaddata(feature &a0, feature &a1, feature &a2)
{
	sz0 = a0.viu.size();
	copying_to_gpu1d(&d_iu,a0.viu);
	copying_to_gpu1d(&d_im,a0.vim);
	copying_to_gpu1d(&d_it,a0.vit);
	copying_to_gpu1d(&d_if,a0.vif);
	copying_to_gpu1d(&d_tb,a0.vtb);
	copying_to_gpu1d(&d_rate,a0.vrate);

	sz1 = a1.viu.size();
	copying_to_gpu1d(&d_iu1,a1.viu);
	copying_to_gpu1d(&d_im1,a1.vim);
	copying_to_gpu1d(&d_it1,a1.vit);
	copying_to_gpu1d(&d_if1,a1.vif);
	copying_to_gpu1d(&d_tb1,a1.vtb);
	copying_to_gpu1d(&d_rate1,a1.vrate);

	sz2 = a2.viu.size();
	copying_to_gpu1d(&d_iu2,a2.viu);
	copying_to_gpu1d(&d_im2,a2.vim);
	copying_to_gpu1d(&d_it2,a2.vit);
	copying_to_gpu1d(&d_if2,a2.vif);
	copying_to_gpu1d(&d_tb2,a2.vtb);
	copying_to_gpu1d(&d_rate2,a2.vrate);

}
float bkmodel_gpu::compute_error()
{
	float sum1 = 0, *dev_sum1;
	hipMalloc((void**) &dev_sum1,sizeof(float));
	hipMemcpy(dev_sum1, &sum1, sizeof(float), hipMemcpyHostToDevice);

	float sum2 = 0, *dev_sum2;
	hipMalloc((void**) &dev_sum2,sizeof(int));
	hipMemcpy(dev_sum2, &sum2, sizeof(float), hipMemcpyHostToDevice);

	kernel_rmse<<<32,32>>>(
	//model
	d_bu,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
 	d_btu,
 	d_bf,
 	d_bt,
 	d_bfm,
 	d_btm,
	//data
	d_iu1,
	d_im1,
	d_it1,
	d_if1,
	d_tb1,
	d_rate1,
	sz1,
	dev_sum1);
	gpuErrchk(hipPeekAtLastError(),400);

	kernel_rmse<<<32,32>>>(
	//model
	d_bu,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
 	d_btu,
 	d_bf,
 	d_bt,
 	d_bfm,
 	d_btm,
	//data
	d_iu2,
	d_im2,
	d_it2,
	d_if2,
	d_tb2,
	d_rate2,
	sz2,
	dev_sum2);
	gpuErrchk(hipPeekAtLastError(),400)
	hipMemcpy( &sum1, dev_sum1,sizeof(float),  hipMemcpyDeviceToHost);
	hipMemcpy( &sum2, dev_sum2,sizeof(float),  hipMemcpyDeviceToHost);
	cout << "in sample error: " << sqrt(sum1/sz1) << endl;
	cout << "out sample error: " <<sqrt(sum2/sz2) << endl;
	float rmse = sqrt(sum2/sz2);

	return rmse;


}

void bkmodel_gpu::test(float lr)
{
	for(int c = 0 ; c<10000; c++)
	{
	kernel_sgd<<<32,32>>>(
	//model
	d_bu,
 	d_bm,
 	d_pm,
 	d_pu,
 	d_pu1,
 	d_btu,
 	d_bf,
 	d_bt,
 	d_bfm,
 	d_btm,
	//data
	d_iu,
	d_im,
	d_it,
	d_if,
	d_tb,
	d_rate,
	lr,
	beg,
	sz0);
	gpuErrchk(hipPeekAtLastError(),300);

	beg= (beg+1024)%sz0;}


/*	testgpu<<<16,16>>>(d_bm,d_bt);
:	hipMemcpy(a,d_bm, 16*sizeof(float),hipMemcpyDeviceToHost);
	cout << a[10] <<"\t"<< a[11] <<"\t"<< a[12] <<endl;
	cout << bm[1][0] << "\t"<< bm[1][1] << "\t" << bm[1][2] << endl;
	gpuErrchk(hipPeekAtLastError());
*/}
